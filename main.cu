#include "hip/hip_runtime.h"
// 670010894879
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include <iostream>
#include <fstream>
#include <string>

struct Pixel
{
  uint8_t blue;
  uint8_t green;
  uint8_t red;
  uint8_t alpha;
};

struct BMP
{
  struct {
    int16_t header;
    uint32_t filesize;
    int16_t reser;
    int16_t reser1;
    uint32_t dataoffset;
  } header;
  struct {
    uint32_t headersize;
    int32_t width;
    int32_t height;
    uint16_t plans;
    uint16_t bpp;
    uint32_t compression;
    uint32_t datasize;
    int32_t re;
    int32_t ve;
    uint32_t color;
    uint32_t importantcolor;
  } info;
  thrust::device_vector<Pixel> data;
};

BMP readBMP(const std::string& filename)
{
    FILE* f = fopen(filename.c_str(), "rb");

    if(f == NULL)
        throw "Argument Exception";

    BMP img;

    fread(&img.header, 14, 1, f);

    fread(&img.info, 40, 1, f);

    thrust::host_vector<Pixel> temp;
    temp.resize(img.info.width * img.info.height);

    fread(&temp[0], sizeof(Pixel), img.info.width * img.info.height, f);

    img.data = thrust::device_vector<Pixel>(temp.begin(), temp.end());

    fclose(f);

    return img;
}

void writeBMP(const std::string& filename, const BMP& image)
{
    FILE* f = fopen(filename.c_str(), "wb");

    fwrite(&image.header, 14, 1, f);

    fwrite(&image.info, 40, 1, f);

    thrust::host_vector<Pixel> temp(image.data.begin(), image.data.end());
    fwrite(&temp[0], sizeof(Pixel), image.data.size(), f); 

    fclose(f);
}

struct reverse_colors
{
  reverse_colors(){}
  __host__ __device__ Pixel operator()(const Pixel& pixel) const { 
      Pixel _pixel;
      _pixel.green = 255 - pixel.green;
      _pixel.red = 255 - pixel.red;
      _pixel.blue = 255 - pixel.blue;
      _pixel.alpha = 255 - pixel.alpha;
      return _pixel;
  }
};

__global__ void increase_contrast(Pixel* A, Pixel* B, size_t rows, size_t cols)
{
  int n = blockDim.x * blockIdx.x + threadIdx.x;

  int i = n / cols;
  int j = n % cols;

  if (i < 0 || j < 0 || i >= rows || j >= cols)
    return;

  if (i > 0 && j > 0 && i < rows - 1 && j < cols - 1)
  {
    const auto val_of_cell = [A, rows, cols](size_t i, size_t j) -> Pixel& {
      return A[i * cols + j];
    };
    
    int green = (int)(val_of_cell(i - 1, j).green) * -1 
               + (int)(val_of_cell(i, j - 1).green) * -1 
               + (int)(val_of_cell(i + 1, j).green) * -1 
               + (int)(val_of_cell(i, j + 1).green) * -1 
               + (int)(val_of_cell(i, j).green) * 5;

    int red = (int)(val_of_cell(i - 1, j).red) * -1 
               + (int)(val_of_cell(i, j - 1).red) * -1 
               + (int)(val_of_cell(i + 1, j).red) * -1 
               + (int)(val_of_cell(i, j + 1).red) * -1 
               + (int)(val_of_cell(i, j).red) * 5; 

    int blue = (int)(val_of_cell(i - 1, j).blue) * -1 
               + (int)(val_of_cell(i, j - 1).blue) * -1 
               + (int)(val_of_cell(i + 1, j).blue) * -1 
               + (int)(val_of_cell(i, j + 1).blue) * -1 
               + (int)(val_of_cell(i, j).blue) * 5; 

    int alpha = (int)(val_of_cell(i - 1, j).alpha) * -1 
               + (int)(val_of_cell(i, j - 1).alpha) * -1 
               + (int)(val_of_cell(i + 1, j).alpha) * -1 
               + (int)(val_of_cell(i, j + 1).alpha) * -1 
               + (int)(val_of_cell(i, j).alpha) * 5; 
    
    B[n].green = green > 255 ? 255 : green;
    B[n].red = red > 255 ? 255 : red;
    B[n].blue = blue > 255 ? 255 : blue;
    B[n].alpha = alpha > 255 ? 255 : alpha;
  }
  else
  {
    B[n] = A[n];
  }
}

int main(int argc, char *argv[])
{
  if (argc == 1)
  {
    std::cout << "Input file name with bmp extension!" << std::endl;
    return;
  }
  std::string name = argv[1];

  auto img = readBMP(name);

  std::cout << "Img was read, data block size - " << img.data.size() << std::endl;

  reverse_colors f;
  thrust::transform(img.data.begin(), img.data.end(), img.data.begin(), f);

  hipDeviceSynchronize();

  std::cout << "Colors were reversed" << std::endl;

  thrust::device_vector<Pixel> copyOfImg = img.data;
  
  Pixel* d_A = thrust::raw_pointer_cast(copyOfImg.data());
  Pixel* d_B = thrust::raw_pointer_cast(img.data.data());

  int threadsPerBlock = 1024;
  int blocksPerGrid = (img.data.size() + threadsPerBlock - 1) / threadsPerBlock;
  increase_contrast<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, img.info.height, img.info.width);
  hipDeviceSynchronize();

  std::cout << hipGetLastError() << std::endl;;

  std::cout << "Contrast was increased" << std::endl;

  writeBMP("./output.bmp", img);
}